#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details. You should have received a copy of the GNU General
   Public License along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*-----------------------------------------------------------------------------------------------100
Dump energy/force/virial with all loaded potentials at a given interval.
--------------------------------------------------------------------------------------------------*/

#include "cavity.cuh"
//#include "nep3_cavity.cuh"
#include "nep3_float.cuh"
//#include "potential_cavity.cuh"
#include "potential_float.cuh"
#include "model/box.cuh"
#include "model/read_xyz.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_vector.cuh"
#include "utilities/read_file.cuh"
#include <iostream>
#include <vector>


const double BOHR_IN_ANGSTROM = 0.529177249;

static __global__ void sum_dipole(
  const int N, const int number_of_patches, const double* g_virial_per_atom, double* g_dipole)
{
  //<<<3, 1024>>>
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  __shared__ double s_d[1024];
  double d = 0.0;

  const unsigned int componentIdx = blockIdx.x * N;

  // 1024 threads, each summing a patch of N/1024 atoms
  for (int patch = 0; patch < number_of_patches; ++patch) {
    int atomIdx = tid + patch * 1024;
    if (atomIdx < N)
      d += g_virial_per_atom[componentIdx + atomIdx];
  }

  // save the sum for this patch
  s_d[tid] = d;
  __syncthreads();

  // aggregate the patches in parallel
  #pragma unroll
  for (int offset = blockDim.x >> 1; offset > 32; offset >>= 1) {
    if (tid < offset) {
      s_d[tid] += s_d[tid + offset];
    }
    __syncthreads();
  }
  for (int offset = 32; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_d[tid] += s_d[tid + offset];
    }
    __syncwarp();
  }

  // save the final value
  if (tid == 0) {
    g_dipole[bid] = s_d[0];
  }
}


static __global__ void sum_dipole_batch(
  const int N,
  const int N_atoms_per_thread,
  const int N_total,
  const double* g_virial_per_atom, 
  double* g_dipole)
{
  // Sums the virials in each of the M=Ntotal/N copies of the system 
  // into [d_x^1,...d_x^M, d_y^1, ..., d_y^M, ...]
  // M is thus the number of system copies, and is equal
  // to the gridDim.y
  // Each thread is responsible for summing N atoms
  //<<<3, M>>>
  
  // We have a 1D thread block of 64 threads
  int tid = threadIdx.x;

  // Each block in the y direction corresponds to
  // a copy of the system.
  int bid = blockIdx.x * gridDim.y + blockIdx.y;
  __shared__ double s_d[64]; // 64 = blockDim.x, since we have 1D thread blocks
  double d = 0.0;

  // Each block sums in x, y and z direction
  const int componentIdx = blockIdx.x * N_total;   // Starting point of the cartesian direction
  const int copyIdx = blockIdx.y * N;              // Start of the current copy of the atoms

  // 64 threads, each summing a patch of N_atoms_per_thread
  for (int patch = 0; patch < N_atoms_per_thread; ++patch) {
    int atomIdx = tid + patch * blockDim.x;
    if (atomIdx < N)
      d += g_virial_per_atom[componentIdx + copyIdx + atomIdx];
  }

  // save the sum for this patch
  s_d[tid] = d;
  __syncthreads();

  // aggregate the patches in parallel
  #pragma unroll
  for (int offset = blockDim.x >> 1; offset > 32; offset >>= 1) {
    if (tid < offset) {
      s_d[tid] += s_d[tid + offset];
    }
    __syncthreads();
  }
  for (int offset = 32; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_d[tid] += s_d[tid + offset];
    }
    __syncwarp();
  }

  // save the final value
  if (tid == 0) {
    g_dipole[bid] = s_d[0];
  }
}


static __global__ void get_center_of_mass(
  const int N, 
  const int number_of_patches, 
  const double total_mass,
  const double* g_mass_per_atom,  
  const double* g_position_per_atom,
  double* g_center_of_mass)
{
  //<<<3, 1024>>>
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  __shared__ double s_d[1024];
  double d = 0.0;

  const unsigned int componentIdx = bid * N;

  // 1024 threads, each summing a patch of N/1024 atoms
  for (int patch = 0; patch < number_of_patches; ++patch) {
    int atomIdx = tid + patch * 1024;
    if (atomIdx < N) {
      d += g_mass_per_atom[atomIdx] * g_position_per_atom[componentIdx + atomIdx];
    }
  }

  // save the sum for this patch
  s_d[tid] = d;
  __syncthreads();

  // aggregate the patches in parallel
  #pragma unroll
  for (int offset = blockDim.x >> 1; offset > 32; offset >>= 1) {
    if (tid < offset) {
      s_d[tid] += s_d[tid + offset];
    }
    __syncthreads();
  }
  for (int offset = 32; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_d[tid] += s_d[tid + offset];
    }
    __syncwarp();
  }

  // save the final value
  if (tid == 0) {
    g_center_of_mass[bid] = s_d[0] / total_mass;
  }
}

static __global__ void setup_copied_systems(
    const int N,
    const int N_atoms_per_system,
    const double* ref_g_pos,
    double* g_pos,
    int* g_index)
{
  // Each atom in the large system of copies will have
  // it's own thread. Depending on it's index, we can
  // figure out which copy it depends on, if it should
  // be a displaced atom, and if so, in what direction.
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
     // Calculate the index for this copy of the smaller system
     unsigned int copyIdx = n1 / N_atoms_per_system;
     g_index[n1] = copyIdx;
     
     // Get the atomIdx from 0-N_atoms_per_system that this
     // thread corresponds to
     unsigned int atomIdx = n1 - copyIdx * N_atoms_per_system;

     // Copy the position for this atom
     g_pos[n1] = ref_g_pos[atomIdx];                                // x position
     g_pos[n1 + N] = ref_g_pos[atomIdx + N_atoms_per_system];       // y position
     g_pos[n1 + 2*N] = ref_g_pos[atomIdx + 2*N_atoms_per_system];   // z position
  }
}

static __global__ void displace_atoms(
    const int N,
    const int N_total,
    const double displacement,
    double* g_pos)
{
  /* Each atom in the smaller atom system has it's own thread.
     For each atom, displace it's corresponding partner in
     the correct copy in the appropriate direction.
     Each atom is displaced in three directions (x,y,z) with
     four different displacements, for a total of 12 displacements
     per atom. There is a total of 4N copies per cartesian direction
     for a total of 12N copies of the system (12N*N atoms in total).

     The displaced systems come as follows:
      i=0, j=0, copyIdx=0: displace atom 0 by +2h in x
      i=0, j=1, copyIdx=1: displace atom 0 by  +h in x
      i=0, j=2, copyIdx=2: displace atom 0 by  -h in x
      i=0, j=3, copyIdx=3: displace atom 0 by -2h in x
      i=0, j=0, copyIdx=4: displace atom 1 by +2h in x
      ...
      i=1, j=0, copyIdx=4N: displace atom 0 by +2h in y
   */
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  const int fourN = 4*N;
  // displacements are done in the order [+2h, +h, -h, -2h]
  const int coefficients[] = {2, 1, -1, -2};
  if (n1 < N) {
    // n1 corresponds to the current atomIdx in the small system
    for (int i = 0; i < 3; i++) {
      for (int j = 0; j < 4; j++) {
        unsigned int copyIdx = i*fourN + j + n1*4;
        unsigned int atomIdx = copyIdx * N + n1; // atomIdx in the large system

        // displace appropriately in the correct direction
        g_pos[atomIdx + i*N_total] += coefficients[j]*displacement;
      }
    }
  }
}

static __global__ void copy_mass_and_type_to_cavity(
  const int N,
  const double* ref_g_mass,
  const int* ref_g_type,
  double* g_mass,
  int* g_type)
{
  // Copy mass and type to the twelve N copies 
  // of the system in AtomCavity
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    for (int i = 0; i < 12*N; i++) {
      g_mass[n1 + i * N] = ref_g_mass[n1];
      g_type[n1 + i * N] = ref_g_type[n1];
    }
  }
}


static __global__ void copy_positions(
  const int N,
  double* ref_g_p,
  double* g_p)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    // Copy the atom positions
    g_p[n1 + 0 * N] = ref_g_p[n1 + 0 * N];
    g_p[n1 + 1 * N] = ref_g_p[n1 + 1 * N];
    g_p[n1 + 2 * N] = ref_g_p[n1 + 2 * N];
  }
}

static __global__ void apply_cavity_force(
  int N,
  double* g_force,
  double* g_cav_force)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    // apply the cavity force to the existing forces
    // from the PES. 
    g_force[n1 + 0 * N] += g_cav_force[n1 + 0 * N];
    g_force[n1 + 1 * N] += g_cav_force[n1 + 1 * N];
    g_force[n1 + 2 * N] += g_cav_force[n1 + 2 * N];
  }
}


static __global__ void initialize_properties(
  int N,
  double* g_pe,
  double* g_f,
  double* g_virial)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    g_pe[n1] = 0.0;
    g_f[n1 + 0 * N] = 0.0;
    g_f[n1 + 1 * N] = 0.0;
    g_f[n1 + 2 * N] = 0.0;
    g_virial[n1 + 0 * N] = 0.0;
    g_virial[n1 + 1 * N] = 0.0;
    g_virial[n1 + 2 * N] = 0.0;
    g_virial[n1 + 3 * N] = 0.0;
    g_virial[n1 + 4 * N] = 0.0;
    g_virial[n1 + 5 * N] = 0.0;
    g_virial[n1 + 6 * N] = 0.0;
    g_virial[n1 + 7 * N] = 0.0;
    g_virial[n1 + 8 * N] = 0.0;
  }
}


Cavity::Cavity(void)
{
  // do nothing
  // This is needed here for some reason
  // due to the NEP3Cavity instance variable.
  // It is probably a complex type that the compiler
  // expects to require a constructor or something.
}

void Cavity::parse(
    const char** param, 
    int num_param, 
    int number_of_atoms)
{
  enabled_ = true;
  printf("Cavity dynamics\n");
  
  FILE* fid_potential = my_fopen(param[1], "r");
  char potential_name[100];
  int count = fscanf(fid_potential, "%s", potential_name);
  if (count != 1) {
    PRINT_INPUT_ERROR("reading error for potential file.");
  }
  // Set up the potential for calculating dipoles with double precision
  number_of_atoms_ = number_of_atoms;
  potential.reset(new NEP3Float(param[1], number_of_atoms));
  potential->N1 = 0;
  potential->N2 = number_of_atoms;
  // and the potential for the jacobian batch calculations
  number_of_copied_systems_ = 12*number_of_atoms_;
  number_of_atoms_in_copied_system_ = number_of_copied_systems_ * number_of_atoms_;
  potential_jacobian.reset(new NEP3Float(param[1], number_of_atoms_in_copied_system_));
  potential_jacobian->N1 = 0;
  potential_jacobian->N2 = number_of_atoms_in_copied_system_;

  if (num_param != 5) {
    PRINT_INPUT_ERROR("cavity should have 4 parameters.");
  }
  if (!is_valid_real(param[2], &coupling_strength)) {
    PRINT_INPUT_ERROR("coupling strength should be a real number.");
  }
  if (coupling_strength < 0.0) {
    PRINT_INPUT_ERROR("coupling strength cannot be negative.");
  }

  if (!is_valid_real(param[3], &cavity_frequency)) {
    PRINT_INPUT_ERROR("cavity frequency should be a real number.");
  }
  if (cavity_frequency < 0.0) {
    PRINT_INPUT_ERROR("cavity frequency cannot be negative.");
  }
  if (!is_valid_int(param[4], &dump_frequency)) {
    PRINT_INPUT_ERROR("dump_frequency should be an integer.");
  }
  printf("   coupling strength %f.\n", coupling_strength);
  printf("   cavity frequency %f.\n", cavity_frequency);
  printf("   dump_frequency %d.\n", dump_frequency);
}

void Cavity::initialize(
Box& box,
Atom& atom,
Force& force)
{
// Setup a dump_exyz with the dump_interval for dump_observer.
if (enabled_) {
  const int number_of_potentials = force.potentials.size();
  std::string jac_filename_ = "jacobian.out";
  std::string cav_filename_ = "cavity.out";
  jacfile_ = my_fopen(jac_filename_.c_str(), "a");
  cavfile_ = my_fopen(cav_filename_.c_str(), "a");
  prevdipole.resize(3);
  cpu_dipole_.resize(3);
  cpu_dipole_jacobian_.resize(number_of_atoms_ * 3 * 3);
  cpu_cavity_force_.resize(number_of_atoms_ * 3);
  gpu_dipole_.resize(3);
  gpu_dipole_jacobian_.resize(number_of_atoms_ * 3 * 3);
  gpu_cavity_force_.resize(number_of_atoms_ * 3);

  // Set up a local copy of the Atoms, on which to compute the dipole
  // Typically in GPUMD we are limited by computational speed, not memory,
  // so we can sacrifice a bit of memory to skip having to recompute the forces
  // & virials with the original potential
  atom_copy.number_of_atoms = number_of_atoms_;
  atom_copy.type.resize(number_of_atoms_);
  atom_copy.mass.resize(number_of_atoms_);
  atom_copy.position_per_atom.resize(number_of_atoms_ * 3);
  atom_copy.force_per_atom.resize(number_of_atoms_ * 3);
  atom_copy.virial_per_atom.resize(number_of_atoms_ * 9);
  atom_copy.potential_per_atom.resize(number_of_atoms_);
  atom_copy.cpu_type.resize(number_of_atoms_);
  atom_copy.cpu_mass.resize(number_of_atoms_);
  atom_copy.cpu_position_per_atom.resize(number_of_atoms_ * 3);

  // Configure the AtomCavity object that will hold all the dipoles
  // for the batched Jacobian calculations
  // This system will have (12*N)*N the number of atoms
  atom_cavity.number_of_atoms = number_of_atoms_in_copied_system_;
  atom_cavity.type.resize(number_of_atoms_in_copied_system_);
  atom_cavity.mass.resize(number_of_atoms_in_copied_system_);
  atom_cavity.position_per_atom.resize(number_of_atoms_in_copied_system_ * 3);
  atom_cavity.force_per_atom.resize(number_of_atoms_in_copied_system_ * 3);
  atom_cavity.virial_per_atom.resize(number_of_atoms_in_copied_system_ * 9);
  atom_cavity.potential_per_atom.resize(number_of_atoms_in_copied_system_);
  atom_cavity.cpu_type.resize(number_of_atoms_in_copied_system_);
  atom_cavity.cpu_mass.resize(number_of_atoms_in_copied_system_);
  atom_cavity.cpu_position_per_atom.resize(number_of_atoms_in_copied_system_ * 3);
  atom_cavity.system_index.resize(number_of_atoms_in_copied_system_);
  atom_cavity.cpu_system_index.resize(number_of_atoms_in_copied_system_);

  // Copy the mass array on atoms to the CPU
  // and compute the total mass. Do this on the CPU
  // since we only need to do it once
  masses_.resize(number_of_atoms_);
  for (int i=0; i<number_of_atoms_; i++) {
    double m_i = atom.cpu_mass[i];
    masses_[i] = m_i;
    mass_ += m_i;
  }
  // Transfer the types and masses to our copy of the Atoms objects
  atom_copy.type.copy_from_host(atom.cpu_type.data());
  atom_copy.mass.copy_from_host(atom.cpu_mass.data());
  // repeat this 12 times for the 12 copies of the system
  // in AtomCavity
  copy_mass_and_type_to_cavity<<<(number_of_atoms_ - 1) / 128 + 1, 128>>>(
    number_of_atoms_,
    atom.mass.data(),
    atom.type.data(),
    atom_cavity.mass.data(),
    atom_cavity.type.data());
  CUDA_CHECK_KERNEL

  // initialize the cavity stuff
  // initial cavity coordinate is equal to
  // self._cav_q0 = self.coupling_strength_v @ dipole_v / self.cavity_frequency
  // so we need the dipole initially

  // TODO clean up
  // Update the dipole and the jacobian
  // we only need the dipole here, but
  // doing one unecessary jacobian calc is
  // not too bad. 
  compute_dipole_and_jacobian(0, box, atom, force);
  // For now, only allow a coupling strength vector in the z-direction.
  // TODO should actually be the charge corrected dipole
  q0 = coupling_strength * cpu_dipole_[2] / cavity_frequency;
  std::cout << "init: " << mass_ << " " << q0  << "\n";

  // set initial values
  cos_integral = 0.0;
    sin_integral = 0.0;
    prevtime = 0.0;
    std::copy(
        cpu_dipole_.begin(),
        cpu_dipole_.end(),
        prevdipole.begin());
  }
}

void Cavity::compute_dipole_and_jacobian(
  int step,
  Box& box,
  Atom& atom,
  Force& force)
{
  if (!enabled_) {
    return;
  }
  // This is probably really bad from a performance perspective
  // and I should remove it and just use the original
  // atoms object.
  // copy positions to the local copy of the atoms object
  copy_positions<<<(number_of_atoms_ - 1) / 128 + 1, 128>>>(
    number_of_atoms_,
    atom.position_per_atom.data(),
    atom_copy.position_per_atom.data());
  CUDA_CHECK_KERNEL

  // Compute the dipole
  // Consider this 
  /* if self.gpumddipole:
      atoms_copy = atoms.copy()
      atoms_copy.set_positions(atoms.get_positions() - atoms.get_center_of_mass())
      gpumd_dipole = (self.calcdipole.get_dipole_moment(atoms_copy) * Bohr +
                      self.charge * atoms.get_center_of_mass())
  */
  get_dipole(box, force);
  gpu_dipole_.copy_to_host(cpu_dipole_.data());
  // The dipole is currently in atomic units.
  // Convert it to the units of the forces, 
  // which are in eV/Å (Bohr -> Å),
  for (int i = 0; i < 3; i++){
    cpu_dipole_[i] *= BOHR_IN_ANGSTROM;
  }
  //std::cout << "Dipole: " << cpu_dipole_[2] << "\n";
  // Compute the dipole jacobian
  // The dipole jacobian has already been converted from atomic
  // units to GPUMD units and shifted appropriately.
  get_dipole_jacobian(box, force, 0.001);
}

void Cavity::compute_and_apply_cavity_force(Atom& atom) {
  if (!enabled_) {
    return;
  }
  // Compute the cavity force
  cavity_force();

  // Apply the cavity force
  // apply the cavity force to the original Atom object,
  // not the local copy. This has the effect of adding 
  // the cavity force on top of the regular PES force.
  gpu_cavity_force_.copy_from_host(cpu_cavity_force_.data());
  apply_cavity_force<<<(number_of_atoms_ - 1) / 128 + 1, 128>>>(
    number_of_atoms_,
    atom.force_per_atom.data(),
    gpu_cavity_force_.data());
  CUDA_CHECK_KERNEL
}

void Cavity::update_cavity(const int step, const double global_time) {
  if (!enabled_) {
    return;
  }
  // Make sure that the frequency is in fs
  // double time = global_time * TIME_UNIT_CONVERSION; // natural (atomic?) units to fs
  double time = global_time; // time in natural units
  // should be done last after atoms have been moved
  // and dipoles and jacobians have been computed
  step_cavity(time);
  
  // Update all properties
  canonical_position(time);
  canonical_momentum(time);
  cavity_potential_energy();
  cavity_kinetic_energy();
}

void Cavity::write(const int step, const double global_time) {
  if (!enabled_) {
    return;
  }
  if ((step + 1) % dump_frequency != 0)
    return;
  // Make sure that the frequency is in fs
  double time = global_time * TIME_UNIT_CONVERSION; // natural (atomic?) units to fs

  // Write properties
  write_dipole(step);
  write_cavity(step, time);
}


void Cavity::get_dipole(
  Box& box,
  Force& force)
{
  initialize_properties<<<(number_of_atoms_ - 1) / 128 + 1, 128>>>(
    number_of_atoms_,
    atom_copy.potential_per_atom.data(),
    atom_copy.force_per_atom.data(),
    atom_copy.virial_per_atom.data());
  CUDA_CHECK_KERNEL
  
  // Reset the dipole
  cpu_dipole_[0] = 0.0;
  cpu_dipole_[1] = 0.0;
  cpu_dipole_[2] = 0.0;
  gpu_dipole_.copy_from_host(cpu_dipole_.data());

  // Compute the dipole
  potential->compute(
    box,
    atom_copy.type,
    atom_copy.position_per_atom,
    atom_copy.potential_per_atom,
    atom_copy.force_per_atom,
    atom_copy.virial_per_atom);
  
  // Aggregate virial_per_atom into dipole
  const int number_of_threads = 1024;
  const int number_of_atoms_per_thread = (number_of_atoms_ - 1) / number_of_threads + 1;
  sum_dipole<<<3, 1024>>>(
    number_of_atoms_,
    number_of_atoms_per_thread,
    atom_copy.virial_per_atom.data(),
    gpu_dipole_.data());
  CUDA_CHECK_KERNEL
}


void Cavity::_get_center_of_mass(GPU_Vector<double>& gpu_center_of_mass) {
  const int number_of_threads = 1024;
  const int number_of_atoms_per_thread = (number_of_atoms_ - 1) / number_of_threads + 1;
  get_center_of_mass<<<3, 1024>>>(
    number_of_atoms_,
    number_of_atoms_per_thread,
    mass_,
    atom_copy.mass.data(),
    atom_copy.position_per_atom.data(),
    gpu_center_of_mass.data());
  CUDA_CHECK_KERNEL
}


void Cavity::get_dipole_jacobian(
  Box& box,
  Force& force,
  double displacement) 
{
  /**
   @brief Get dipole gradient through finite differences.
   @details Calculates the dipole gradient, a (N_atoms, 3, 3) tensor for the
   gradients dµ_k/dr_ij, for atom i, Cartesian direction j (x, y, z) and dipole
   moment component k.
   Before computing the gradient the dipoles are corrected using the center of
   mass and the total system charge, supplied via the parameter `charge`.
   @param displacement        Displacement in Å.
   @param charge              Total system charge, used to correct dipoles.
  */
  const int N_cartesian = 3;
  const int N_components = 3;
  const int values_per_atom = N_cartesian * N_components;
  const int BLOCK_SIZE = 128;
  
  // Second order central differences
  // Need to compute four dipoles for each structure, yielding an error O(h^4)
  // Coefficients are defined here:
  // https://en.wikipedia.org/wiki/Finite_difference_coefficient#Central_finite_difference

  const double one_over_displacement =
      1.0 / displacement; // coefficients are scaled properly
  const double c0 = -1.0 / 12.0; // coefficient for 2h
  const double c1 = 2.0 / 3.0;   // coefficient for h
  

  int values_per_direction = 4 * number_of_atoms_;
  const int number_of_copies = 3 * 4 * number_of_atoms_;

  // Step 1: Setup the 12N cavity atom system for batched
  // calculation of all dipoles
  setup_copied_systems<<<(number_of_atoms_in_copied_system_ - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(
    number_of_atoms_in_copied_system_,
    number_of_atoms_,
    atom_copy.position_per_atom.data(),
    atom_cavity.position_per_atom.data(),
    atom_cavity.system_index.data());
  CUDA_CHECK_KERNEL
  
  displace_atoms<<<(number_of_atoms_ - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(
    number_of_atoms_,
    number_of_atoms_in_copied_system_,
    displacement,
    atom_cavity.position_per_atom.data());
  CUDA_CHECK_KERNEL

  // Step 2: Compute the dipoles in the batched system
  initialize_properties<<<(number_of_atoms_in_copied_system_ - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(
    number_of_atoms_in_copied_system_,
    atom_cavity.potential_per_atom.data(),
    atom_cavity.force_per_atom.data(),
    atom_cavity.virial_per_atom.data());
  CUDA_CHECK_KERNEL
  
  // Compute the dipole
  potential_jacobian->compute_jacobian(
    box,
    number_of_copies,
    atom_cavity.type,
    atom_cavity.position_per_atom,
    atom_cavity.potential_per_atom,
    atom_cavity.force_per_atom,
    atom_cavity.virial_per_atom,
    atom_cavity.system_index);

  std::vector<double> cpu_virial_per_atom_small(number_of_atoms_*9);
  std::vector<double> cpu_virial_per_atom_large(number_of_atoms_in_copied_system_*9);
  atom_cavity.virial_per_atom.copy_to_host(cpu_virial_per_atom_large.data());
  atom_copy.virial_per_atom.copy_to_host(cpu_virial_per_atom_small.data());

  
  // Step 3: Collect all dipoles
  const int number_of_threads = 64;
  const int number_of_atoms_per_thread = (number_of_atoms_ - 1) / number_of_threads + 1;
  // The systems we study are typically small, so we'll use
  // a block size of 64 threads. Each block will sum the dipole
  // in one copy of the system, with one thread summing number_of_atoms_ / 64 atoms.
  // The blocks will then be launched in grids of size (3, number_of_copies),
  // where 3 corresponds to the number of cartesian directions.
  // Thus, each block will have access to a contigous chunk of memory
  // corresponding to the values for a certain copy of the system.
  GPU_Vector<double> gpu_dipole_batch(3 * number_of_copies);
  std::vector<double> cpu_dipole_batch(3 * number_of_copies);
  dim3 gridDim(3, number_of_copies);
  sum_dipole_batch<<<gridDim, number_of_threads>>>(
    number_of_atoms_,
    number_of_atoms_per_thread,
    number_of_atoms_in_copied_system_,
    atom_cavity.virial_per_atom.data(),
    gpu_dipole_batch.data());
  CUDA_CHECK_KERNEL
  gpu_dipole_batch.copy_to_host(cpu_dipole_batch.data());

  // Check the dipoles
  // std::cout << "-----\n";
  // std::cout << "Ref: " << cpu_dipole_[2] << "\n";
  // for (int i=0; i<number_of_copies; i++) {
  //   std::cout << i << ": " << cpu_dipole_batch[i + number_of_copies*2]*BOHR_IN_ANGSTROM << "\n";
  // }
  // std::cout << "-----\n";

  // Step 4: Compute the jacobian
  // For now we skip the charge correction
  // Each thread 
  //sum_dipoles_into_jacobian<<<gridDim, number_of_threads>>>(
  //  number_of_atoms_,
  //  number_of_atoms_per_thread,
  //  number_of_atoms_in_copied_system_,
  //  atom_cavity.virial_per_atom.data(),
  //  gpu_dipoles.data());
  //CUDA_CHECK_KERNEL

  for (int i = 0; i < N_cartesian; i++) {
    for (int j = 0; j < number_of_atoms_; j++) {
      // index of the current group of four displacements
      // dipoles come in the order [d_x^1,d_x^2,d_x^3,d_x^4,...,d_x^M, d_y^1,..., d_z^M]
      int group_of_four_copies_index = values_per_direction * i + j * 4;

      for (int k = 0; k < N_components; k++) {
        int componentIdx = k * number_of_copies;
        double dipole_forward_two_h = cpu_dipole_batch[componentIdx + group_of_four_copies_index + 0];
        double dipole_forward_one_h = cpu_dipole_batch[componentIdx + group_of_four_copies_index + 1];
        double dipole_backward_one_h = cpu_dipole_batch[componentIdx + group_of_four_copies_index + 2];
        double dipole_backward_two_h = cpu_dipole_batch[componentIdx + group_of_four_copies_index + 3];
        cpu_dipole_jacobian_[i * N_components + j * values_per_atom + k] =
            (c0 * (dipole_forward_two_h * BOHR_IN_ANGSTROM) +
             c1 * (dipole_forward_one_h * BOHR_IN_ANGSTROM) -
             c1 * (dipole_backward_one_h * BOHR_IN_ANGSTROM)-
             c0 * (dipole_backward_two_h * BOHR_IN_ANGSTROM)) *
            one_over_displacement;
      }
    }
  }
}

void Cavity::canonical_position(const double time) {
  /* 
    Cavity position coordinate

        q(t) = sin(ω(t-t₀)) Icos - cos(ω(t-t₀)) Isin + q(t₀) cos(ω(t-t₀))

    where

                t
        Icos = ∫  dt' cos(ωt') λ⋅μ
                t₀

    and

                t
        Isin = ∫  dt' sin(ωt') λ⋅μ
                t₀
    
  */
  double phase = cavity_frequency * time;
  q = sin(phase) * cos_integral
      - cos(phase) * sin_integral
      + q0 * cos(phase);
}

void Cavity::canonical_momentum(const double time) {
  /*
      Cavity momentum coordinate

      p(t) = ω cos(ω(t-t₀)) Icos + ω sin(ω(t-t₀)) Isin - q(t₀) ω sin(ω(t-t₀))

      where

              t
      Icos = ∫  dt' cos(ωt') λ⋅μ
              t₀

      and

              t
      Isin = ∫  dt' sin(ωt') λ⋅μ
              t₀
  */
  double phase = cavity_frequency * time;
  p = cavity_frequency * (
      cos(phase) * cos_integral
      + sin(phase) * sin_integral
      - q0 * sin(phase));
}


void Cavity::cavity_potential_energy() {
  /*
     Potential energy of the cavity
        0.5 (ω q(t) - λ⋅μ(t))²
  */
  // For now, only allow a coupling strength vector in the z-direction.
  double coup_times_dip = coupling_strength * cpu_dipole_[2];
  double cav_factor = cavity_frequency * q - coup_times_dip;
  cavity_pot = 0.5 * cav_factor * cav_factor;
}


void Cavity::cavity_kinetic_energy() {
  /*
     Kinetic energy of the cavity
       0.5 p(t)²
  */
  cavity_kin = 0.5 * p * p;
}


void Cavity::cavity_force() {
  /* Force from the cavity 
     get_dipole, get_dipole_jacobian and
     step() should have been run before
     this function.

     This function can be replaced with a kernel
     once the jacobian is no longer the time limiting step.
   */

  // initialize the cavity force
  for (int i = 0; i < 3*number_of_atoms_; i++){
    cpu_cavity_force_[i] = 0.0;
  }

  // njdip_iv = dipole_jacobian_ivv @ self.coupling_strength_v
  // force_iv = njdip_iv * (self.cavity_frequency * self.canonical_position
  //                           - self.coupling_strength_v @ dipole_v)
  double cav_factor = cavity_frequency * q - coupling_strength*cpu_dipole_[2];
  int N_components = 3;
  int values_per_atom = 9;
  for (int j = 0; j < number_of_atoms_; j++){

    // The coupling is non-zero only in the z-direction
    // so we only need to grap the k=2 components. 
    // the jacobian is indexed as
    // [i * N_components + j * values_per_atom + k]
    cpu_cavity_force_[j + 0*number_of_atoms_] = cav_factor*coupling_strength*cpu_dipole_jacobian_[0 * N_components + j * values_per_atom + 2];
    cpu_cavity_force_[j + 1*number_of_atoms_] = cav_factor*coupling_strength*cpu_dipole_jacobian_[1 * N_components + j * values_per_atom + 2];
    cpu_cavity_force_[j + 2*number_of_atoms_] = cav_factor*coupling_strength*cpu_dipole_jacobian_[2 * N_components + j * values_per_atom + 2];
  }
}

void Cavity::step_cavity(double time) {
  /*
    Step the time dependent potential by time dt.
    Should be called after updating the positions
  */
  // TODO
  double dt = time - prevtime;
  double prevlmu = coupling_strength * prevdipole[2];
  double lmu = coupling_strength * cpu_dipole_[2];

  
  // std::cout << time << " " << dt << " " << prevlmu << " " << lmu << " " << cavity_frequency << " " << prevdipole[2] << " " << cpu_dipole_[2] << "\n";
  cos_integral += 0.5 * dt * cos(cavity_frequency * prevtime) * prevlmu;
  sin_integral += 0.5 * dt * sin(cavity_frequency * prevtime) * prevlmu;
  cos_integral += 0.5 * dt * cos(cavity_frequency * time) * lmu;
  sin_integral += 0.5 * dt * sin(cavity_frequency * time) * lmu;

  // Copy current values to previous
  prevtime = time;
  std::copy(
      cpu_dipole_.begin(),
      cpu_dipole_.end(),
      prevdipole.begin());
}


void Cavity::write_dipole(const int step)
{
  // stress components are in Voigt notation: xx, yy, zz, yz, xz, xy
  fprintf(jacfile_, "%d%20.10e%20.10e%20.10e", step, cpu_dipole_[0], cpu_dipole_[1], cpu_dipole_[2]);
  // for (int i = 0; i < cpu_dipole_jacobian_.size(); i++) {
  //   fprintf(jacfile_, "%20.10e", cpu_dipole_jacobian_[i]);
  // }
  fprintf(jacfile_, "\n");
  fflush(jacfile_);
}

void Cavity::write_cavity(const int step, const double time)
{
  // stress components are in Voigt notation: xx, yy, zz, yz, xz, xy
  fprintf(cavfile_, "%d%20.10e%20.10e%20.10e%20.10e%20.10e%20.10e%20.10e", step, time, q, p, cavity_pot, cavity_kin, cos_integral, sin_integral);
  for (int i = 0; i < cpu_cavity_force_.size(); i++) {
    fprintf(cavfile_, "%20.10e", cpu_cavity_force_[i]);
  }
  fprintf(cavfile_, "\n");
  fflush(cavfile_);
}

void Cavity::finalize()
{
  if (enabled_) {
    fclose(jacfile_);
    fclose(cavfile_);
    enabled_ = false;
  }
}
